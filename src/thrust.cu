#include "hip/hip_runtime.h"
#include <thrust/version.h>
#include <thrust/device_vector.h>

#include <Rcpp.h>

// [[Rcpp::export]]
void thrust_version(void) {
  int major = THRUST_MAJOR_VERSION;
  int minor = THRUST_MINOR_VERSION;
  int subminor = THRUST_SUBMINOR_VERSION;
  int patch = THRUST_PATCH_NUMBER;
  
  Rcpp::Rcout << "Thrust v" << major << "." << minor << "."
              << subminor << "-" << patch << "\n";
              
}

struct sq_exp_cov_functor{
  double sigma2;
  double range;
  sq_exp_cov_functor(double s, double r) : sigma2(s), range(r) { }
  __host__ __device__ double operator()(double &d) const{
    return sigma2 * exp(-d*d * range*range);
  }
};

Rcpp::NumericMatrix gpu_sq_exp_cov(Rcpp::NumericMatrix const& d, 
                                   double sigma2 = 1, double range=1) {
  thrust::device_vector<double> D(d.begin(), d.end());
  thrust::transform(D.begin(), D.end(), D.begin(), sq_exp_cov_functor(sigma2, range));
  Rcpp::NumericMatrix cov(d.nrow(), d.ncol());
  thrust::copy(D.begin(), D.end(), cov.begin());
  return cov;
}